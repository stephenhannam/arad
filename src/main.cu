#include <iostream>

#include "gpu.h"

// function to add the elements of two arrays
void addArraysCPU(double *x, double *y, int numElements)
{
	for (int i = 0; i < numElements; i++)
	{
		y[i] = x[i] + y[i];
	}
}


int main()
{
	std::cout << "Hello, world!" << std::endl;

	printCudaVersion();

	// Code to show how to add two arrays on the GPU 

	int N = 10;

	double *x_h = new double[N];
  	double *y_h = new double[N];

	// initialize x and y arrays on the host
	for (int i = 0; i < N; i++) {
		x_h[i] = 1.0;
		y_h[i] = 2.0;
	}

	// Allocate Unified Memory – accessible from CPU or GPU
	double *x_d;
  	double *y_d;
	hipMalloc(&x_d, N*sizeof(double));
	hipMalloc(&y_d, N*sizeof(double));

	// Copy data from host to device array
	hipMemcpy(x_d, x_h, N*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(y_d, y_h, N*sizeof(double), hipMemcpyHostToDevice);

	// Run kernel on 1M elements on the GPU
	addArraysGPU(x_d, y_d, N, 0);

	// Wait for GPU to finish before accessing on host
	hipDeviceSynchronize();

	// Copy values back to host arrays
	hipMemcpy(x_h, x_d, N*sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(y_h, y_d, N*sizeof(double), hipMemcpyDeviceToHost);

	// Check for errors (all values should be 3.0f)
	for (int i = 0; i < N; i++)
	{
		std::cout << y_h[i] << std::endl;
	}

	// Free host memory
	free(x_h);
	free(y_h);

	// Free device memory
	hipFree(x_d);
	hipFree(y_d);
  


	return 0;
}

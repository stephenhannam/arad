#include "hip/hip_runtime.h"
#include <iostream>
#include "gpu.h"

void printCudaVersion()
{
	std::cout << "CUDA Compiled version: " << __CUDACC_VER__ << std::endl;

	int runtime_ver;
	hipRuntimeGetVersion(&runtime_ver);
	std::cout << "CUDA Runtime version: " << runtime_ver << std::endl;

	int driver_ver;
	hipDriverGetVersion(&driver_ver);
	std::cout << "CUDA Driver version: " << driver_ver << std::endl;
}

/** 
 * @brief Sample Kernel function to add the elements of two arrays on the GPU. Each element
 * of x and y will be added together, with the result being stored in y
 * 
 * @param x first input array
 * @param y second input array
 * @param numElements number of elements in array x and y
 */
__global__ void addArraysKernel(double *x, double *y, int numElements)
{
	const auto idx = blockIdx.x * blockDim.x + threadIdx.x;
	
	if(idx < numElements)
	{
		y[idx] = x[idx] + y[idx];
	}
}

void addArraysGPU(double *x, double *y, int numElements, hipStream_t stream)
{
	const int threads = 1024;
	const int blocks = (numElements + threads - 1) / threads;
	const int sharedMemory = 0;

	addArraysKernel<<<blocks, threads, sharedMemory, stream>>>(x, y, numElements);
}
